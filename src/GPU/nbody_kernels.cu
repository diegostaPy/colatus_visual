
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <math_functions.h>
#include <stdio.h>

#define TILE_WIDTH 64
#define SOFTENING_SQUARED 0.001

#define __USE_SHARED__
//#define __USE_NBODY_CALC__


// WRAP is used to force each block to start working on a different 
// chunk (and wrap around back to the beginning of the array) so that
// not all multiprocessors try to read the same memory locations at 
// once.
#define WRAP(x,m) (((x)<m)?(x):(x-m))  // Mod without divide, works on values from 0 up to 2m



namespace NBody {
    namespace Kernels {
        

        
        
        __global__ void show(float4* position,
                                        float4* pos) {

            int bx     = blockIdx.x;
            int tx     = threadIdx.x;
            int dimX   = blockDim.x;
            int idx    = bx * dimX + tx;
            float4 pos_v = pos[idx];

            position[idx]= pos_v;



        }
    }
}


   
extern "C" {
    void g_show(float4* position, float4* pos,int numbodies) {
        // Processamento do delta
        dim3 dimBlock(TILE_WIDTH, 1, 1);
        dim3 dimGrid((numbodies/dimBlock.x), 1);

        NBody::Kernels::show<<<dimGrid, dimBlock>>>(position, pos);


    }
}




